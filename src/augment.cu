#include "hip/hip_runtime.h"
#include "augment.h"
#include <stdexcept>



template <typename in_t, typename out_t>
__global__ void dataugPaddingKernel(const in_t* in, out_t* out, size_t inWidth, size_t height, size_t outWidth) {
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= inWidth || y >= height)
        return;

    unsigned int i = 3 * ((blockIdx.z * height + y) * inWidth + x);
    unsigned int o = 4 * ((blockIdx.z * height + y) * outWidth + x);

    out[o + 0] = in[i + 0];
    out[o + 1] = in[i + 1];
    out[o + 2] = in[i + 2];
}


__global__ void dataugProcessingKernel(hipTextureObject_t texObj, float* out, const size_t width, const size_t height, const size_t batchSize, const Params* params) {
    // get pixel position
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= width || y >= height)
        return;

    const auto& imgParams = params[blockIdx.z];

    // calculate normalized texture coordinates
    float u = ((float)x + 0.5f) / width;
    float v = ((float)y + 0.5f) / height;

    // apply translation
    u += imgParams.translation[0];
    v += imgParams.translation[1];

    // apply rotation
    u -= 0.5f;
    v -= 0.5f;
    float z = u * imgParams.geom[2][0] + v * imgParams.geom[2][1] + 1;
    float tu = (u * imgParams.geom[0][0] + v * imgParams.geom[0][1]) / z + 0.5f;
    float tv = (u * imgParams.geom[1][0] + v * imgParams.geom[1][1]) / z + 0.5f;

    // apply flipping
    if (imgParams.flags & FLAG_HORIZONTAL_FLIP)
        tu = 1.0f - tu;
    if (imgParams.flags & FLAG_VERTICAL_FLIP)
        tv = 1.0f - tv;

    // unroll V to the batch
    float tv_ = (blockIdx.z + __saturatef(tv)) / batchSize;

    // sample the input texture
    float4 sample = tex2D<float4>(texObj, tu, tv_);

    // get another sample (Mixup)
    if (blockIdx.z != imgParams.mixImgIdx) {
        if (imgParams.flags & FLAG_MIX_HORIZONTAL_FLIP)
            tu = 1.0f - tu;
        if (imgParams.flags & FLAG_MIX_VERTICAL_FLIP)
            tv = 1.0f - tv;

        tv_ = (imgParams.mixImgIdx + __saturatef(tv)) / batchSize;
        float4 sample2 = tex2D<float4>(texObj, tu, tv_);

        sample.x = (1 - imgParams.mixFactor) * sample.x + imgParams.mixFactor * sample2.x;
        sample.y = (1 - imgParams.mixFactor) * sample.y + imgParams.mixFactor * sample2.y;
        sample.z = (1 - imgParams.mixFactor) * sample.z + imgParams.mixFactor * sample2.z;
    }

    // fill surroundings
    if (tu <= 0.0f || tu >= 1.0f || tv <= 0.0f || tv >= 1.0f)
        sample.x = sample.y = sample.z = 0.5f;

    // cutout
    if (imgParams.flags & FLAG_CUTOUT) {
        if (abs(tu - imgParams.cutoutPos[0]) < imgParams.cutoutSize[0] && abs(tv - imgParams.cutoutPos[1]) < imgParams.cutoutSize[1])
            sample.x = sample.y = sample.z = 0.5f;
    }

    // apply color transform and rotate
    unsigned int i = 3 * ((blockIdx.z * height + y) * width + x);
    out[i    ] = __saturatef(imgParams.color[0][0] * sample.x + imgParams.color[0][1] * sample.y + imgParams.color[0][2] * sample.z);
    out[i + 1] = __saturatef(imgParams.color[1][0] * sample.x + imgParams.color[1][1] * sample.y + imgParams.color[1][2] * sample.z);
    out[i + 2] = __saturatef(imgParams.color[2][0] * sample.x + imgParams.color[2][1] * sample.y + imgParams.color[2][2] * sample.z);
}


void padChannels(hipStream_t stream, const uint8_t* input, uint8_t* output, size_t width, size_t height, size_t batchSize, size_t outWidth) {
    const dim3 threads(32, 32);
    const dim3 blocks((width  + threads.x - 1) / threads.x,
                      (height + threads.y - 1) / threads.y,
                      batchSize);

    dataugPaddingKernel<uint8_t, uint8_t> <<<blocks, threads, 0, stream>>> (input, output, width, height, outWidth);
}


void compute(hipStream_t stream, const uint8_t* input, float* output, size_t inWidth, size_t inHeight, size_t pitch, size_t outWidth, size_t outHeight, size_t batchSize, const Params* params) {
    // set up texture
    struct hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypePitch2D;
    resDesc.res.pitch2D.devPtr = const_cast<uint8_t*>(input);
    resDesc.res.pitch2D.desc.f = hipChannelFormatKindUnsigned;
    resDesc.res.pitch2D.desc.w = 8;
    resDesc.res.pitch2D.desc.x = 8;
    resDesc.res.pitch2D.desc.y = 8;
    resDesc.res.pitch2D.desc.z = 8;
    resDesc.res.pitch2D.width = inWidth;
    resDesc.res.pitch2D.height = inHeight * batchSize;
    resDesc.res.pitch2D.pitchInBytes = pitch;

    struct hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.addressMode[0]   = hipAddressModeClamp;
    texDesc.addressMode[1]   = hipAddressModeClamp;
    texDesc.filterMode       = hipFilterModeLinear;
    texDesc.readMode         = hipReadModeNormalizedFloat;
    texDesc.normalizedCoords = 1;

    hipTextureObject_t texObj = 0;
    auto error = hipCreateTextureObject(&texObj, &resDesc, &texDesc, nullptr);
    if (error != hipSuccess)
        throw std::runtime_error("Cannot create texture object: " + std::string(hipGetErrorString(error)));

    // run kernel
    const dim3 threads(32, 32);
    const dim3 blocks(
        (outWidth  + threads.x - 1) / threads.x,
        (outHeight + threads.y - 1) / threads.y,
        batchSize
    );

    dataugProcessingKernel<<<blocks, threads, 0, stream>>>(texObj, output, outWidth, outHeight, batchSize, params);

    // destroy texture
    hipDestroyTextureObject(texObj);

    // check for errors
    error = hipGetLastError();
    if (error != hipSuccess)
        throw std::runtime_error(hipGetErrorString(error));
}


void setColorTransform(Params& params, float hueShiftRad, float saturationFactor, float valueFactor) {
    // Sampling a rotation and scaling matrix in RGB space:
    //   - rotation around (1,1,1) vector by hueShiftRad radians,
    //   - scaling along (1,1,1) vector by valueFactor and in orthogonal direction by saturationFactor
    static const float sqrt3 = sqrtf(3);
    const float
        c = cosf(hueShiftRad),
        s = sinf(hueShiftRad);
    const float
        _1 = (valueFactor * (12 * saturationFactor * c + 6)) / 18,
        _2 = (valueFactor * (6 * saturationFactor * c + 6 * sqrt3 * saturationFactor * s - 6)) / 18,
        _3 = (valueFactor * (6 * sqrt3 * saturationFactor * s - 6 * saturationFactor * c + 6)) / 18;

    params.color[0][0] = _1;
    params.color[0][1] = -_2;
    params.color[0][2] = _3;

    params.color[1][0] = _3;
    params.color[1][1] = _1;
    params.color[1][2] = -(valueFactor * (saturationFactor * c + sqrt3 * saturationFactor * s - 1)) / 3;

    params.color[2][0] = -_2;
    params.color[2][1] = _3;
    params.color[2][2] = (valueFactor * (4 * saturationFactor * c + 2)) / 6;
}


void setGeometricTransform(Params& params, float pan, float tilt, float roll, float scaleX, float scaleY) {
    /*
        X, Y: image axes, Z: forward

        Rotation in XZ plane (tilt):
            [ cos(a)  0  sin(a)
                   0  1       0
             -sin(a)  0  cos(a) ]

        Rotation in YZ plane (pan):
            [ 1        0      0
              0   cos(b) sin(b)
              0  -sin(b) cos(b) ]

        Rotation in XY plane (roll):
            [  cos(c) sin(c)  0
              -sin(c) cos(c)  0
                    0      0  1 ]

        Considering the image is on Z=0 plane, the camera is at Z=1 point
    */

    const float
        cosA = std::cos(pan), sinA = std::sin(pan),
        cosB = std::cos(tilt), sinB = std::sin(tilt),
        cosC = std::cos(roll), sinC = std::sin(roll);

    params.geom[0][0] = -sinA * sinB * sinC + cosA * cosC;
    params.geom[0][1] = sinC * cosB;
    params.geom[0][2] =  sinA * cosC + sinB * sinC * cosA;

    params.geom[1][0] = -sinA * sinB * cosC - sinC * cosA;
    params.geom[1][1] = cosB * cosC;
    params.geom[1][2] = -sinA * sinC + sinB * cosA * cosC;

    // apply scaling factors
    params.geom[0][0] *= scaleX;
    params.geom[0][1] *= scaleX;

    params.geom[1][0] *= scaleY;
    params.geom[1][1] *= scaleY;
}